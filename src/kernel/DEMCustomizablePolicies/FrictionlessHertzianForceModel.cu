#include "hip/hip_runtime.h"
// DEM force calculation strategies, modifiable

if (overlapDepth > 0) {
    // Material properties
    float E_cnt, CoR_cnt;
    {
        matProxy2ContactParam<float>(E_cnt, E[bodyAMatType], nu[bodyAMatType], E[bodyBMatType], nu[bodyBMatType]);
        // CoR is pair-wise, so obtain it this way
        CoR_cnt = CoR[bodyAMatType][bodyBMatType];
    }

    float3 rotVelCPA, rotVelCPB;
    {
        // We also need the relative velocity between A and B in global frame to use in the damping terms
        // To get that, we need contact points' rotational velocity in GLOBAL frame
        // This is local rotational velocity (the portion of linear vel contributed by rotation)
        rotVelCPA = cross(ARotVel, locCPA);
        rotVelCPB = cross(BRotVel, locCPB);
        // This is mapping from local rotational velocity to global
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPA.x, rotVelCPA.y, rotVelCPA.z, AOriQ.w, AOriQ.x, AOriQ.y,
                                                AOriQ.z);
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPB.x, rotVelCPB.y, rotVelCPB.z, BOriQ.w, BOriQ.x, BOriQ.y,
                                                BOriQ.z);
    }

    // The (total) relative linear velocity of A relative to B
    const float3 velB2A = (ALinVel + rotVelCPA) - (BLinVel + rotVelCPB);
    const float projection = dot(velB2A, B2A);

    const float mass_eff = (AOwnerMass * BOwnerMass) / (AOwnerMass + BOwnerMass);
    float sqrt_Rd = sqrt(overlapDepth * (ARadius * BRadius) / (ARadius + BRadius));
    const float Sn = 2. * E_cnt * sqrt_Rd;

    const float loge = (CoR_cnt < DEME_TINY_FLOAT) ? log(DEME_TINY_FLOAT) : log(CoR_cnt);
    float beta = loge / sqrt(loge * loge + deme::PI_SQUARED);

    const float k_n = deme::TWO_OVER_THREE * Sn;
    const float gamma_n = deme::TWO_TIMES_SQRT_FIVE_OVER_SIX * beta * sqrt(Sn * mass_eff);

    // normal force (that A feels)
    force += (k_n * overlapDepth + gamma_n * projection) * B2A;

    // printf("A linear vel is (%.9g, %.9g, %.9g)\n", ALinVel.x, ALinVel.y, ALinVel.z);
    // printf("A rotational vel is (%.9g, %.9g, %.9g)\n", ARotVel.x, ARotVel.y, ARotVel.z);
    // printf("locCPA is (%.9g, %.9g, %.9g)\n", locCPA.x, locCPA.y, locCPA.z);
    // printf("Force is (%.9g, %.9g, %.9g) on body %d\n", force.x, force.y, force.z, AOwner);
    // printf("CoR_cnt is %.9g\n", CoR_cnt);
    // printf("Sn is %.9g, sqrt_Rd is %.9g\n", Sn, sqrt_Rd);
    // printf("k_n is %.9g, gamma_n is %.9g\n", k_n, gamma_n);
    // printf("overlapDepth is %.9g, projection is %.9g\n", overlapDepth, projection);
}
